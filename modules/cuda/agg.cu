#include "hip/hip_runtime.h"
#define THREADS _THREADS_

__device__ void _increment_hit_count(
    const int grid_size,
    const int i,
    const int j,
    const bool *grid,
    const float *massx,
    const float *massy,
    const int *neigh,
    int *hits
    ){
  const float one = 1.0f/(float)grid_size;
  const int k = i*grid_size+j;

  int ii;
  int jj;
  float x;
  float y;

  for (int w=1;w<2;w++){
    x = (float)i*one+massx[k]*one*w;
    y = (float)j*one+massy[k]*one*w;
    ii = (int)round(x*grid_size);
    jj = (int)round(y*grid_size);
    atomicAdd(&hits[ii*grid_size+jj], 1);
  }

  return;
}

__global__ void agg(
    const int n,
    const int grid_size,
    const bool *grid,
    const float *massx,
    const float *massy,
    const int *neigh,
    int *hits
    ){
  const int ij = blockIdx.x*THREADS + threadIdx.x;
  const int i = (int)floor(float(ij)/(float)grid_size);
  const int j = (ij-grid_size*i);

  if (ij>=n){
    return;
  }
  if (neigh[ij]<1){
    return;
  }
  if (!grid[ij]){
    return;
  }

  _increment_hit_count(grid_size, i, j, grid, massx, massy, neigh, hits);
}
